#include"hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>

using namespace std;

__global__ void matrixMul(float *a,float *b,float *c,int size)
{
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int column = blockIdx.x*blockDim.x+threadIdx.x;
	int sum=0;
	for(int i=0;i<size;i++)
	{
		sum+=a[row*size+i]*b[i*size+column];
	}
	c[row*size+column]=sum;
}

void multiply(float *h_a,float *h_b,float *h_c,int row_per_proc,int n,int id,float *gpu_time)
{
	
	cout<<endl<<endl;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float *d_a,*d_b,*d_r;
	hipSetDevice(id);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,id);
	cout<<"Device["<<id<<"] "<<deviceProp.name<<endl;
	hipEventRecord(start,0);
	hipMalloc(&d_a,row_per_proc*n*sizeof(float));
        hipMalloc(&d_b,n*n*sizeof(float));
	hipMalloc(&d_r,row_per_proc*n*sizeof(float));
	
	hipMemcpy(d_a,h_a,row_per_proc*n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,n*n*sizeof(float),hipMemcpyHostToDevice);
	
	cout<<endl<<endl;
	int threads_per_block =16;
	dim3 block_size(threads_per_block,threads_per_block);
	dim3 grid_size(n/block_size.x,row_per_proc/block_size.y);

	matrixMul<<<grid_size,block_size>>>(d_a,d_b,d_r,n);
	hipEventRecord(stop,0);
	hipMemcpy(h_c,d_r,row_per_proc*n*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_r);
	hipEventElapsedTime(gpu_time,start,stop);
}
